﻿/**
* ARQUITECTURA DE COMPUTADORES
* 2º Grado en Ingenieria Informatica
*
* Entrega 4
*
* Alumno: Rodrigo Pascual Arnaiz y Villar Solla, Alejandro
* Fecha: 11/11/2022
*
*/
///////////////////////////////////////////////////////////////////////////
// includes
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

///////////////////////////////////////////////////////////////////////////
// defines
///////////////////////////////////////////////////////////////////////////

// declaracion de funciones
// HOST: funciones llamadas desde el host y ejecutada en el host

/**
* Funcion: propiedadesDispositivo
* Objetivo: Mustra las propiedades del dispositvo, esta funcion
* es ejecutada llamada y ejecutada desde el host
*
* Param: INT id_dispositivo -> ID del dispotivo
* Return: cudaDeviceProp -> retorna el onjeto que tiene todas las
* propiedades del dispositivo CUDA
*/
__host__ hipDeviceProp_t propiedadesDispositivo(int id_dispositivo)
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, id_dispositivo);
	// calculo del numero de cores (SP)
	int cuda_cores = 0;
	int multi_processor_count = deviceProp.multiProcessorCount;
	int major = deviceProp.major;
	int minor = deviceProp.minor;
	char* arquitectura = (char*)"";
	switch (major)
	{
	case 1:
		//TESLA
		cuda_cores = 8;
		arquitectura = (char*)"TESLA";
		break;
	case 2:
		//FERMI
		arquitectura = (char*)"FERMI";
		if (minor == 0)
			cuda_cores = 32;
		else
			cuda_cores = 48;
		break;
	case 3:
		//KEPLER
		arquitectura = (char*)"KEPLER";
		cuda_cores = 192;
		break;
	case 5:
		//MAXWELL
		arquitectura = (char*)"MAXWELL";
		cuda_cores = 128;
		break;
	case 6:
		//PASCAL
		arquitectura = (char*)"PASCAL";
		cuda_cores = 64;
		break;
	case 7:
		//VOLTA
		arquitectura = (char*)"VOLTA";
		cuda_cores = 64;
		break;
	case 8:
		//AMPERE
		arquitectura = (char*)"AMPERE";
		cuda_cores = 128;
		break;
	default:
		arquitectura = (char*)"DESCONOCIDA";
		//DESCONOCIDA
		cuda_cores = 0;
		printf("!!!!!dispositivo desconocido!!!!!\n");
	}
	// presentacion de propiedades
	printf("***************************************************\n");
	printf("DEVICE %d: %s\n", id_dispositivo, deviceProp.name);
	printf("***************************************************\n");
	printf("> Capacidad de Computo \t\t\t: %d.%d\n", major, minor);
	printf("> Arquitectura CUDA \t\t\t: %s \n", arquitectura);
	printf("> No. de MultiProcesadores \t\t: %d \n",
		multi_processor_count);
	printf("> No. de CUDA Cores (%dx%d) \t\t: %d \n", cuda_cores,
		multi_processor_count, cuda_cores*
		multi_processor_count);
	printf("> No. max. de Hilos (por bloque) \t: %d \n",
		deviceProp.maxThreadsPerBlock);
	printf("***************************************************\n");

	return deviceProp;
}

/**
* Funcion: mostrarArray
* Mostrar el valor que tiene un array de una dirección
*
* @param int *array -> Array a mostrar
* @param int tam -> Tamaño del array
*/
__host__ void mostrarArray(int* array, int tam)
{
	for (int i = 0; i < tam; i++)
	{
		printf("%i ", array[i]);
	}
	
}


/**
* Funcion: pedirUnNumero
* Pedir un numero entre un rango
*
* @param char* texto -> Texto a mostrar
* @param int minimo -> Número minimo
* @param int maximo -> Número maximo
* @return int -> Número instroducido por el usuario
*/
__host__ int pedirUnNumero(char* texto, int minimo, int maximo)
{
	// Comenzamos en con el valor minimo menos uno pra que siempre se muestre una vez
	int resultado = minimo -1 ;

	// Preguntamos hasta obtener un valor correcto
	while (resultado < minimo || resultado > maximo)
	{
		// Mostramos el texto por pantalla
		printf("%s: ", texto);

		// Leemos el valor introducido por un usuario
		scanf("%i", &resultado);

		// En el caso de que el numero del usuario supere al numero minimo 
		if (resultado < minimo)
		{
			printf("\nEl valor introducido no puede ser menor de %i", minimo);
		}

		//  En el caso de que el numero del usuario supere al numero maximo
		if (resultado > maximo)
		{
			printf("\nEl valor introducido no puede ser mayor de %i", maximo);
		}

	}
	return resultado;
}

/**
* Funcion: rellenarArrayAleatorio
* Llenar el array de forma alezatoria
*
* @param int *arr  -> Array que hay que rellenar 
* @param int tam -> Tamaño del array
*/
__host__ void rellenarArrayAleatorio(int* arr, int tam)
{
	for (int i = 0; i < tam; i++)
	{
		// El numero tiene que estar entre 1 y 31 
		arr[i] = (int)(rand() % 30 + 1 );
	}
}


// declaracion de funciones
// DEVICE

/**
* Funcion: ordenacionPorRango
* Función que ordena un array utilizando el algoritmo de ordenacion por rango
*
* @param int* original -> array original que hay que ordenar 
* @param int* final -> Array ordenado que devuelve el valor
* @param int size -> Tamaño del array 
*/
__global__ void ordenacionPorRango(int* original, int* final, int tam)
{
	int rango = 0;
	for (int i = 0; i < tam; i++)
	{
		if (original[threadIdx.x] > original[i])
		{
			rango++;
		}
		if (original[threadIdx.x] == original[i] && threadIdx.x > i)
		{
			rango++;
		}
		final[rango] = original[threadIdx.x];
	}
}

///////////////////////////////////////////////////////////////////////////
// MAIN: Inicio del programa
int main(int argc, char** argv)
{
	// Para que cada ejecucion sea aleatoria 
	srand(time(NULL));

	// Declaración de variables
	int* hst_original, * hst_final; // Array en el hots 
	int* dev_original, * dev_final; // Array en el device 
	int tamanyo, numero_dispositivos; // Tamño del los arrays y numero de dispositivos de ejecucion CUDA 
	float elapsedTime; // Elipsis de tiempo 
	hipDeviceProp_t props; // Guardar propiedades
	// Declaración de eventos
	hipEvent_t inicio;
	hipEvent_t fin;

	// buscando dispositivos
	hipGetDeviceCount(&numero_dispositivos);

	if (numero_dispositivos != 0)
	{
		for (int i = 0; i < numero_dispositivos; i++)
		{
			props = propiedadesDispositivo(i);
		}
	}
	else
	{
		printf("!!!!!ERROR!!!!!\n");
		printf("Este ordenador no tiene dispositivo de ejecucion CUDA\n");
		printf("<pulsa [INTRO] para finalizar>");
		getchar();
		return 1;
	}

	
	// Creación de eventos
	hipEventCreate(&inicio);
	hipEventCreate(&fin);

	// Preguntar por el numero de elementos a ordenar 
	tamanyo = pedirUnNumero("\nElige una cantidad de elementos para el vector", 0, props.maxThreadsPerBlock);

	// Asignación de espacio a las variables en el host
	hst_original = (int*)malloc(tamanyo * sizeof(int));
	hst_final = (int*)malloc(tamanyo * sizeof(int));

	// Asignación de espacio a las variables en el device
	hipMalloc((void**)&dev_original, tamanyo * sizeof(int));
	hipMalloc((void**)&dev_final, tamanyo * sizeof(int));

	// Llenar array con números aleatorios entre 1 y 31
	rellenarArrayAleatorio(hst_original, tamanyo);

	// Copiar datos al dispositivo
	hipMemcpy(dev_original, hst_original, sizeof(int) * tamanyo,hipMemcpyHostToDevice);

	
	hipEventRecord(inicio, 0); // Iniciamos el evento de inicio a 0
	ordenacionPorRango <<<1, tamanyo >>> (dev_original, dev_final, tamanyo); 	// Ordenar por rango
	hipEventRecord(fin, 0); // Iniciamos el evento de fin a 0

	// Sincronizar Eventos
	hipEventSynchronize(fin);

	// Traer datos del device
	hipMemcpy(hst_final, dev_final, sizeof(int) * tamanyo,hipMemcpyDeviceToHost);
	// Calcular tiempo de ejecucion
	hipEventElapsedTime(&elapsedTime, inicio, fin);

	// Mostrar tiempo de ejecución por pantalla
	printf("> Kernel de %i bloque con %i hilos (%i hilos)\n", 1, tamanyo, 1* tamanyo);
	printf("> Tiempo Ejecucion:\t%f ms\n", elapsedTime);


	// Mostrar arrays
	printf("> VECTOR INCIAL:\n");
	mostrarArray(hst_original, tamanyo);
	printf("\n> VECTOR ORDENADO:\n");
	mostrarArray(hst_final, tamanyo);
	printf("\n");

	// Destruimos los eventos
	hipEventDestroy(inicio);
	hipEventDestroy(fin);

	printf("***************************************************\n");
	printf("<pulsa [INTRO] para finalizar>");
	getchar();
	return 0;



} 
